#include "hip/hip_runtime.h"
#include "deform_conv_cuda_kernel.h"

#include <cstdio>

#define CUDA_KERNEL_LOOP(i, n)                                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);                 \
       i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;

inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename DType>
__device__ DType deformable_im2col_bilinear(const DType *bottom_data,
                                            const int data_width,
                                            const int height, const int width,
                                            DType h, DType w) {

  int h_low = floor(h);
  int w_low = floor(w);
  int h_high;
  int w_high;
  if (h_low >= height - 1) {
    h_high = h_low = height - 1;
    h = (DType)h_low;
  } else {
    h_high = h_low + 1;
  }

  if (w_low >= width - 1) {
    w_high = w_low = width - 1;
    w = (DType)w_low;
  } else {
    w_high = w_low + 1;
  }

  DType lh = h - h_low;
  DType lw = w - w_low;
  DType hh = 1 - lh, hw = 1 - lw;

  DType v1 = bottom_data[h_low * data_width + w_low];
  DType v2 = bottom_data[h_low * data_width + w_high];
  DType v3 = bottom_data[h_high * data_width + w_low];
  DType v4 = bottom_data[h_high * data_width + w_high];
  DType w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  DType val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename DType>
__device__ DType get_gradient_weight(DType argmax_h, DType argmax_w,
                                     const int h, const int w, const int height,
                                     const int width) {

  if (argmax_h < 0 || argmax_h > height || argmax_w < 0 || argmax_w > width) {
    // empty
    return 0;
  }

  argmax_h = max(argmax_h, (DType)0.0f);
  argmax_w = max(argmax_w, (DType)0.0f);

  int argmax_h_low = (int)argmax_h;
  int argmax_w_low = (int)argmax_w;
  int argmax_h_high;
  int argmax_w_high;
  if (argmax_h_low >= height - 1) {
    argmax_h_high = argmax_h_low = height - 1;
    argmax_h = (DType)argmax_h_low;
  } else {
    argmax_h_high = argmax_h_low + 1;
  }
  if (argmax_w_low >= width - 1) {
    argmax_w_high = argmax_w_low = width - 1;
    argmax_w = (DType)argmax_w_low;
  } else {
    argmax_w_high = argmax_w_low + 1;
  }
  DType weight = 0;
  if (h == argmax_h_low) {
    if (w == argmax_w_low) {
      weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
    } else if (w == argmax_w_high) {
      weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
    }
  } else if (h == argmax_h_high) {
    if (w == argmax_w_low) {
      weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
    } else if (w == argmax_w_high) {
      weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
    }
  }
  return weight;
}

template <typename DType>
__device__ DType get_coordinate_weight(DType argmax_h, DType argmax_w,
                                       const int height, const int width,
                                       const DType *im_data,
                                       const int data_width, const int bp_dir) {

  if (argmax_h < 0 || argmax_h > height || argmax_w < 0 || argmax_w > width) {
    // empty
    return 0;
  }

  if (argmax_h < 0)
    argmax_h = 0;
  if (argmax_w < 0)
    argmax_w = 0;

  int argmax_h_low = (int)argmax_h;
  int argmax_w_low = (int)argmax_w;
  int argmax_h_high;
  int argmax_w_high;
  if (argmax_h_low >= height - 1) {
    argmax_h_high = argmax_h_low = height - 1;
    argmax_h = (DType)argmax_h_low;
  } else {
    argmax_h_high = argmax_h_low + 1;
  }
  if (argmax_w_low >= width - 1) {
    argmax_w_high = argmax_w_low = width - 1;
    argmax_w = (DType)argmax_w_low;
  } else {
    argmax_w_high = argmax_w_low + 1;
  }
  DType weight = 0;

  if (bp_dir == 0) {
    weight += -1 * (argmax_w_low + 1 - argmax_w) *
              im_data[argmax_h_low * data_width + argmax_w_low];
    weight += -1 * (argmax_w - argmax_w_low) *
              im_data[argmax_h_low * data_width + argmax_w_high];
    weight += (argmax_w_low + 1 - argmax_w) *
              im_data[argmax_h_high * data_width + argmax_w_low];
    weight += (argmax_w - argmax_w_low) *
              im_data[argmax_h_high * data_width + argmax_w_high];
  } else if (bp_dir == 1) {
    weight += -1 * (argmax_h_low + 1 - argmax_h) *
              im_data[argmax_h_low * data_width + argmax_w_low];
    weight += (argmax_h_low + 1 - argmax_h) *
              im_data[argmax_h_low * data_width + argmax_w_high];
    weight += -1 * (argmax_h - argmax_h_low) *
              im_data[argmax_h_high * data_width + argmax_w_low];
    weight += (argmax_h - argmax_h_low) *
              im_data[argmax_h_high * data_width + argmax_w_high];
  }

  return weight;
}

template <typename DType>
__global__ void deformable_im2col_gpu_kernel(
    const int n, const DType *data_im, const DType *data_offset,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int channel_per_deformable_group, const int height_col,
    const int width_col, DType *data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    // index index of output matrix
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int c_im = (index / width_col) / height_col;
    const int c_col = c_im * kernel_h * kernel_w;

    // compute deformable group index
    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;
    DType *data_col_ptr =
        data_col + (c_col * height_col + h_col) * width_col + w_col;
    const DType *data_im_ptr = data_im + (c_im * height + h_in) * width + w_in;
    const DType *data_offset_ptr = data_offset + deformable_group_index * 2 *
                                                     kernel_h * kernel_w *
                                                     height_col * width_col;

    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        const int data_offset_h_ptr =
            ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr =
            ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col +
            w_col;
        const DType offset_h = data_offset_ptr[data_offset_h_ptr];
        const DType offset_w = data_offset_ptr[data_offset_w_ptr];
        DType val = static_cast<DType>(0);
        const DType h_im = h_in + i * dilation_h + offset_h;
        const DType w_im = w_in + j * dilation_w + offset_w;
        if (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) {
          const DType map_h = i * dilation_h + offset_h;
          const DType map_w = j * dilation_w + offset_w;
          const int cur_height = height - h_in;
          const int cur_width = width - w_in;
          val = deformable_im2col_bilinear(data_im_ptr, width, cur_height,
                                           cur_width, map_h, map_w);
        }
        *data_col_ptr = val;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

template <typename DType>
void deformable_im2col(hipStream_t stream, const DType *data_im,
                       const DType *data_offset, const int channels,
                       const int height, const int width, const int ksize_h,
                       const int ksize_w, const int pad_h, const int pad_w,
                       const int stride_h, const int stride_w,
                       const int dilation_h, const int dilation_w,
                       const int deformable_group, DType *data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  int channel_per_deformable_group = channels / deformable_group;
  // Launch
  deformable_im2col_gpu_kernel<<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0,
                                 stream>>>(
      num_kernels, data_im, data_offset, height, width, ksize_h, ksize_w, pad_h,
      pad_w, stride_h, stride_w, dilation_h, dilation_w,
      channel_per_deformable_group, height_col, width_col, data_col);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in deformable_im2col: %s\n", hipGetErrorString(err));
    // TODO(BZ) panic
  }
}

template void deformable_im2col<float>(
    hipStream_t stream, const float *data_im, const float *data_offset,
    const int channels, const int height, const int width, const int ksize_h,
    const int ksize_w, const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int dilation_h, const int dilation_w,
    const int deformable_group, float *data_col);

template <typename DType>
__global__ void deformable_col2im_gpu_kernel(
    const int n, const DType *data_col, const DType *data_offset,
    const int channels, const int height, const int width, const int kernel_h,
    const int kernel_w, const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int dilation_h, const int dilation_w,
    const int channel_per_deformable_group, const int height_col,
    const int width_col, DType *grad_im) {
  CUDA_KERNEL_LOOP(index, n) {
    const int j = (index / width_col / height_col) % kernel_w;
    const int i = (index / width_col / height_col / kernel_w) % kernel_h;
    const int c = index / width_col / height_col / kernel_w / kernel_h;
    // compute the start and end of the output

    const int deformable_group_index = c / channel_per_deformable_group;

    int w_out = index % width_col;
    int h_out = (index / width_col) % height_col;
    int w_in = w_out * stride_w - pad_w;
    int h_in = h_out * stride_h - pad_h;

    const DType *data_offset_ptr = data_offset + deformable_group_index * 2 *
                                                     kernel_h * kernel_w *
                                                     height_col * width_col;
    const int data_offset_h_ptr =
        ((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out;
    const int data_offset_w_ptr =
        ((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col + w_out;
    const DType offset_h = data_offset_ptr[data_offset_h_ptr];
    const DType offset_w = data_offset_ptr[data_offset_w_ptr];
    const DType cur_inv_h_data = h_in + i * dilation_h + offset_h;
    const DType cur_inv_w_data = w_in + j * dilation_w + offset_w;

    const DType cur_top_grad = data_col[index];
    const int cur_h = (int)cur_inv_h_data;
    const int cur_w = (int)cur_inv_w_data;
    for (int dy = -2; dy <= 2; dy++) {
      for (int dx = -2; dx <= 2; dx++) {
        if (cur_h + dy >= 0 && cur_h + dy < height && cur_w + dx >= 0 &&
            cur_w + dx < width && abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            abs(cur_inv_w_data - (cur_w + dx)) < 1) {
          int cur_bottom_grad_pos =
              (c * height + cur_h + dy) * width + cur_w + dx;
          DType weight =
              get_gradient_weight(cur_inv_h_data, cur_inv_w_data, cur_h + dy,
                                  cur_w + dx, height, width);
          atomicAdd(grad_im + cur_bottom_grad_pos, weight * cur_top_grad);
        }
      }
    }
  }
}

template <typename DType>
void deformable_col2im(hipStream_t stream, const DType *data_col,
                       const DType *data_offset, const int channels,
                       const int height, const int width, const int ksize_h,
                       const int ksize_w, const int pad_h, const int pad_w,
                       const int stride_h, const int stride_w,
                       const int dilation_h, const int dilation_w,
                       const int deformable_group, DType *grad_im) {

  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * ksize_h * ksize_w * height_col * width_col;
  int channel_per_deformable_group = channels / deformable_group;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  deformable_col2im_gpu_kernel<<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS, 0,
                                 stream>>>(
      num_kernels, data_col, data_offset, channels, height, width, ksize_h,
      ksize_w, pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
      channel_per_deformable_group, height_col, width_col, grad_im);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in deformable_col2im: %s\n", hipGetErrorString(err));
    // TODO(BZ) panic
  }
}

template void deformable_col2im<float>(
    hipStream_t stream, const float *data_col, const float *data_offset,
    const int channels, const int height, const int width, const int ksize_h,
    const int ksize_w, const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int dilation_h, const int dilation_w,
    const int deformable_group, float *grad_im);

template <typename DType>
__global__ void deformable_col2im_coord_gpu_kernel(
    const int n, const DType *data_col, const DType *data_im,
    const DType *data_offset, const int channels, const int height,
    const int width, const int kernel_h, const int kernel_w, const int pad_h,
    const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int channel_per_deformable_group, const int height_col,
    const int width_col, DType *grad_offset) {
  CUDA_KERNEL_LOOP(index, n) {
    DType val = 0;
    int w = index % width_col;
    int h = (index / width_col) % height_col;
    int c = index / width_col / height_col;
    // compute the start and end of the output

    const int deformable_group_index = c / (2 * kernel_h * kernel_w);
    const int col_step = kernel_h * kernel_w;
    int cnt = 0;
    const DType *data_col_ptr = data_col + deformable_group_index *
                                               channel_per_deformable_group *
                                               width_col * height_col;
    const DType *data_im_ptr =
        data_im + deformable_group_index * channel_per_deformable_group /
                      kernel_h / kernel_w * height * width;
    const DType *data_offset_ptr = data_offset + deformable_group_index * 2 *
                                                     kernel_h * kernel_w *
                                                     height_col * width_col;

    const int offset_c = c - deformable_group_index * 2 * kernel_h * kernel_w;

    for (int col_c = (offset_c / 2); col_c < channel_per_deformable_group;
         col_c += col_step) {
      const int col_pos = ((col_c * height_col) + h) * width_col + w;
      const int bp_dir = offset_c % 2;

      int j = (col_pos / width_col / height_col) % kernel_w;
      int i = (col_pos / width_col / height_col / kernel_w) % kernel_h;
      int w_out = col_pos % width_col;
      int h_out = (col_pos / width_col) % height_col;
      int w_in = w_out * stride_w - pad_w;
      int h_in = h_out * stride_h - pad_h;
      const int data_offset_h_ptr =
          (((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out);
      const int data_offset_w_ptr =
          (((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col +
           w_out);
      const DType offset_h = data_offset_ptr[data_offset_h_ptr];
      const DType offset_w = data_offset_ptr[data_offset_w_ptr];
      DType inv_h = h_in + i * dilation_h + offset_h;
      DType inv_w = w_in + j * dilation_w + offset_w;
      if (inv_h < 0 || inv_w < 0 || inv_h >= height || inv_w >= width) {
        inv_h = inv_w = -1;
      }
      const DType weight = get_coordinate_weight(
          inv_h, inv_w, height, width, data_im_ptr + cnt * height * width,
          width, bp_dir);
      val += weight * data_col_ptr[col_pos];
      cnt += 1;
    }

    grad_offset[index] = val;
  }
}

template <typename DType>
void deformable_col2im_coord(hipStream_t stream, const DType *data_col,
                             const DType *data_im, const DType *data_offset,
                             const int channels, const int height,
                             const int width, const int ksize_h,
                             const int ksize_w, const int pad_h,
                             const int pad_w, const int stride_h,
                             const int stride_w, const int dilation_h,
                             const int dilation_w, const int deformable_group,
                             DType *grad_offset) {

  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels =
      height_col * width_col * 2 * ksize_h * ksize_w * deformable_group;
  int channel_per_deformable_group =
      channels * ksize_h * ksize_w / deformable_group;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  deformable_col2im_coord_gpu_kernel<<<GET_BLOCKS(num_kernels),
                                       CUDA_NUM_THREADS, 0, stream>>>(
      num_kernels, data_col, data_im, data_offset, channels, height, width,
      ksize_h, ksize_w, pad_h, pad_w, stride_h, stride_w, dilation_h,
      dilation_w, channel_per_deformable_group, height_col, width_col,
      grad_offset);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in deformable_col2im: %s\n", hipGetErrorString(err));
    // TODO(BZ) panic
  }
}

template void
deformable_col2im_coord(hipStream_t stream, const float *data_col,
                        const float *data_im, const float *data_offset,
                        const int channels, const int height, const int width,
                        const int ksize_h, const int ksize_w, const int pad_h,
                        const int pad_w, const int stride_h, const int stride_w,
                        const int dilation_h, const int dilation_w,
                        const int deformable_group, float *grad_offset);
